

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

__global__  void offsetCopy(float* A, float* B, int offset) 
{ 
    // threadIdx.x is a built-in variable provided by CUDA at runtime 
    long int i = blockIdx.x * blockDim.x + threadIdx.x + offset; 
    A[i] = B[i];
}


int  main(int argc, char **argv)
{
    unsigned long int N = 16777216;  // default vector size
    int offset = 0;     // default offset
    float *devPtrA;
    float *devPtrB; 

    if (argc > 1) N = atoi(argv[1]) / 4;  // get size of the vectors
    if (argc > 2) offset = atoi(argv[2]);  // get size of the vectors

    printf("Running GPU misaligned copy for %u bytes and offset %d\n", N * sizeof(float), offset);

    hipSetDevice(0);

    hipMalloc((void**)&devPtrA, (N+offset) * sizeof(float)); 
    hipMalloc((void**)&devPtrB, (N+offset) * sizeof(float)); 

    hipEvent_t start1, stop1, start2, stop2;
    float time, k_time;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    // figure out time to start a kernel
    hipEventRecord(start1, 0);
    offsetCopy<<<1, 1>>>(devPtrA, devPtrB, offset);
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&k_time, start1, stop1);

    // do the bandwidth test
    hipEventRecord(start2, 0);

    offsetCopy<<<N/512, 512>>>(devPtrA, devPtrB, offset);

    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);

    // check for errors
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) 
    {
        fprintf(stderr, "CUDA error: %s.\n", hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }

    hipEventElapsedTime(&time, start2, stop2);

    hipEventDestroy(start1);
    hipEventDestroy(stop1);

    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    hipFree(devPtrA); 
    hipFree(devPtrB);    

    printf("effective bandwidth %f GBytes/sec\n", 2.0f * N * sizeof(float) / ((time-k_time) / 1000) / 1e9);
}

