

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

__global__  void strideCopy(float* A, float* B, int stride) 
{ 
    // threadIdx.x is a built-in variable provided by CUDA at runtime 
    long int i = (blockIdx.x * blockDim.x + threadIdx.x) * stride; 
    A[i] = B[i];
}


int  main(int argc, char **argv)
{
    unsigned long int N = 16777216;  // default vector size
    int stride = 1;     // default offset
    float *devPtrA;
    float *devPtrB; 

    if (argc > 1) N = atoi(argv[1]) / 4;  // get size of the vectors
    if (argc > 2) stride = atoi(argv[2]);  // get size of the vectors

    printf("Running GPU misaligned copy for %u bytes and stride %d\n", N * sizeof(float), stride);

    hipSetDevice(0);

    hipMalloc((void**)&devPtrA, (N*stride) * sizeof(float)); 
    hipMalloc((void**)&devPtrB, (N*stride) * sizeof(float)); 

    hipEvent_t start1, stop1, start2, stop2;
    float time, k_time;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    // figure out time to start a kernel
    hipEventRecord(start1, 0);
    strideCopy<<<1, 1>>>(devPtrA, devPtrB, stride);
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&k_time, start1, stop1);

    // do the bandwidth test
    hipEventRecord(start2, 0);

    strideCopy<<<N/512, 512>>>(devPtrA, devPtrB, stride);

    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);

    // check for errors
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) 
    {
        fprintf(stderr, "CUDA error: %s.\n", hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }

    hipEventElapsedTime(&time, start2, stop2);

    hipEventDestroy(start1);
    hipEventDestroy(stop1);

    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    hipFree(devPtrA); 
    hipFree(devPtrB);    

    printf("effective bandwidth %f GBytes/sec\n", 2.0f * N * sizeof(float) / ((time-k_time) / 1000) / 1e9);
}

